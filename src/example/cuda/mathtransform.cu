
#include <hip/hip_runtime.h>
#define ADDITION 0
#define SUBTRACTION 1
#define MULTIPLICATION 2
#define DIVISION 3






__global__ void mathTransform(float* globalBuffer, int globalSize, int type, int amount)
{
   __shared__ float localValue;

   int i = blockIdx.x * blockDim.x + threadIdx.x;

   if ( i >= globalSize )
   {
      return;
   }

   localValue = globalBuffer[i];
   switch (type)
   {
   case ADDITION:
      localValue += amount;
      break;
   case SUBTRACTION:
      localValue -= amount;
      break;
   case MULTIPLICATION:
      localValue *= amount;
      break;
   case DIVISION:
      localValue /= amount;
      break;
   }
   globalBuffer[i] = localValue;
}
